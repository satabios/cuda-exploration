
#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCK_DIM 4
#define GRID_DIM 2

__global__ void reduce_kernel(float* input, float* partialSums, unsigned int N) {
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2 + tid;

    printf("Block %d, thread %d, i %d, Input:%f \n ", blockIdx.x, tid, i, input[i]);


    // Each thread performs a reduction within a block
    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        if (tid % (2 * stride) == 0 && i + stride < N) {
            input[i] += input[i + stride];
        }
        __syncthreads(); // Synchronize threads within the block
    }

    // Write the reduced value to partial sums
    if (tid == 0) {
        partialSums[blockIdx.x] = input[i];
    }
}

int main() {
    const unsigned int N = 4 * 4;  // Size of input array
    float* input = (float*)malloc(N * sizeof(float));
    float* partialSums = (float*)malloc(GRID_DIM * sizeof(float));

    // Initialize the input array with random values
    for (unsigned int i = 0; i < N; i++) {
        input[i] = i;
    }

    // Allocate device memory
    float* d_input;
    float* d_partialSums;
    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_partialSums, GRID_DIM * sizeof(float));

    // Copy input data to the device
    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch the reduction kernel
    reduce_kernel<<<GRID_DIM, BLOCK_DIM>>>(d_input, d_partialSums, N);

    // Copy the partial sums back to the host
    hipMemcpy(partialSums, d_partialSums, GRID_DIM * sizeof(float), hipMemcpyDeviceToHost);

    // Perform a final reduction on the partial sums on the host
    float finalSum = 0.0f;
    for (unsigned int i = 0; i < GRID_DIM; i++) {
        finalSum += partialSums[i];
    }

    // Print the result
    printf("Final sum: %f\n", finalSum);

    // Free memory
    free(input);
    free(partialSums);
    hipFree(d_input);
    hipFree(d_partialSums);

    return 0;
}
